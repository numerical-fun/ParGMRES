#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_ptr.h>
#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/print.h>
#include <cusp/multiply.h>

#include "gmres.cuh"

template <class Matrix>
static csr_mat_t get_csr_mat_t(Matrix& A){

    csr_mat_t csr_mat;

    csr_mat.nrow = A.num_rows;
    csr_mat.ncol = A.num_cols;
    csr_mat.nnz = A.values.size();
    csr_mat.value = thrust::raw_pointer_cast(A.values.data());
    csr_mat.cindex = thrust::raw_pointer_cast(A.column_indices.data());
    csr_mat.rowstart = thrust::raw_pointer_cast(A.row_offsets.data());

    return csr_mat;
}

template <class Vector>
static vec_t get_vec_t(Vector& x){

    vec_t vec_x;

    vec_x.value = thrust::raw_pointer_cast(x.data());
    vec_x.size = x.size();
    
    return vec_x;
}

static void test_matrix_setup(){
    
    // create an empty sparse matrix structure (CSR format)
    cusp::csr_matrix<int, float, cusp::device_memory> A;

    // load a matrix stored in MatrixMarket format
    cusp::io::read_matrix_market_file(A, "../../data/cage4.mtx");

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<float, cusp::device_memory> x(A.num_cols);
    cusp::array1d<float, cusp::device_memory> b(A.num_rows, 1);

    // get raw pointer
    csr_mat_t csr_mati = get_csr_mat_t(A);
    vec_t vec_x = get_vec_t(x);
    vec_t vec_b = get_vec_t(b);
}

static void test_s_x_sqrt(){
    
    cusp::array1d<float, cusp::device_memory> x(10, 9);
    cusp::array1d<float, cusp::device_memory> y(10);
    
    float *p_x = thrust::raw_pointer_cast(x.data());
    float *p_y = thrust::raw_pointer_cast(y.data());
    
    s_x_sqrt<<<1, 256>>>(p_y, p_x, 10);
    
    // it should be all 3s
    cusp::print(y);
}

static void test_s_x_div_a(){
 
    cusp::array1d<float, cusp::device_memory> x(10, 9);
    cusp::array1d<float, cusp::device_memory> y(10);
    cusp::array1d<float, cusp::device_memory> a(1, 2);
    
    float *p_x = thrust::raw_pointer_cast(x.data());
    float *p_y = thrust::raw_pointer_cast(y.data());
    float *p_a = thrust::raw_pointer_cast(a.data());
    
    s_x_div_a<<<1, 256>>>(p_y, p_x, p_a, 10);

    // it should be all 4.5s
    cusp::print(y);
}


static void test_s_x_dot_y(){
 
    cusp::array1d<float, cusp::device_memory> x(10);
    cusp::array1d<float, cusp::device_memory> y(10);
    cusp::array1d<float, cusp::device_memory> res(1);
    
    for(int i = 0; i < 10; i++){
        x[i] = i+1;
        y[i] = i+1;
    }
    
    float *p_x = thrust::raw_pointer_cast(x.data());
    float *p_y = thrust::raw_pointer_cast(y.data());
    float *p_res = thrust::raw_pointer_cast(res.data());
    
    s_x_dot_y<<<1, 256>>>(p_res, p_y, p_x, 10);

    // it should be 91
    cusp::print(res);
}

static void test_s_x_sub_ay(){

    // void s_x_sub_ay(float *x, float *y, float *a, int N){    
    
    cusp::array1d<float, cusp::device_memory> x(10, 9);
    cusp::array1d<float, cusp::device_memory> y(10, 2);
    cusp::array1d<float, cusp::device_memory> a(1, 2);
    
    float *p_x = thrust::raw_pointer_cast(x.data());
    float *p_y = thrust::raw_pointer_cast(y.data());
    float *p_a = thrust::raw_pointer_cast(a.data());

    s_x_sub_ay<<<1, 256>>>(p_x, p_y, p_a, 10);

    // it should be 5
    cusp::print(x);
}

static void test_s_mat_mul_x(){

    // create an empty sparse matrix structure (CSR format)
    cusp::csr_matrix<int, float, cusp::device_memory> A;

    // load a matrix stored in MatrixMarket format
    cusp::io::read_matrix_market_file(A, "../../data/cage4.mtx");

    cusp::array1d<float, cusp::device_memory> x(A.num_cols, 1);
    cusp::array1d<float, cusp::device_memory> y1(A.num_rows);
    cusp::array1d<float, cusp::device_memory> y2(A.num_rows);

    // get raw pointer
    csr_mat_t mat = get_csr_mat_t(A);
    float *p_x = thrust::raw_pointer_cast(x.data());
    float *p_y1 = thrust::raw_pointer_cast(y1.data());

    s_mat_mul_x<<<1, 256>>>(p_y1, mat, p_x);
    
    // y1 should be the same as y2
    cusp::print(y1);
    
    cusp::multiply(A, x, y2);

    cusp::print(y2);
}

static void test_gmres_update_x(){
    //(float *x, float *V, float *y, int m, int N); 

}

static void test_gmres_compute_r0(){
    //(float *r0, csr_mat_t mat, float *x, vec_t vec, float *beta);

}

int main(){

    std::cout << "\nTest program:\n\n";

    test_s_x_sqrt();
    test_s_x_div_a();
    test_s_x_dot_y();
    test_s_x_sub_ay();

    return 0;
}


